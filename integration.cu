#include"stdio.h"
#include"stdlib.h"
#include<math.h>
#include <sys/time.h>    
#define USECUDA 1
#if USECUDA==1
#include<hip/hip_runtime.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#endif
#define Dev_Loop 1024
#define BlockN 1024
#define addNum 16
//这个程序由CUDA并行构架编写而成
long getCurrentTime()  
{  
	struct timeval tv;  
	gettimeofday(&tv,NULL);  
	return tv.tv_sec * 1000000 + tv.tv_usec;  
}
double randomf()//c 的随机数产生器改为产生小数
{
	return ((double)rand())/RAND_MAX;
}
#if USECUDA==1
//以下代码全部在CUDA构架下运行的
__global__ void inte_cell(double* l,double* r,double *res,long *time,hiprandState *state)
{
	//不要传函数指针
	int i = blockIdx.x*BlockN+threadIdx.x;
	long seed=(*time)+(i);//因为所有给定时间一定，所以我们只能通过对时间进行简单处理
	int offset=0;//完全独立的序列，所以offset全部为零来节约时间
	hiprand_init (seed,i,offset,&state[i]);//设置第i个随机序列
	double x=1,sum=0;
	double k=8;
	for(int j=0;j<k*Dev_Loop;j++)
	{
		x=(r[i]-l[i])*hiprand_uniform_double(&state[i]);
		sum+=sqrt(x+sqrt(x));//func(x);
	}
	res[i]=sum/(Dev_Loop*k);
	__syncthreads();

}
__global__ void big_plus(double*a,double *res,int *threadNum)
{
	//为了尽可能的利用并行效率，加法采用两次树形相加的形式，每次加addNum个
	//如此可以对付2^30次的快速相加
	//虽然嘛。。。。这是毫无意义的啦！因为本程序只有2^20次的相加
	//不过留个接口以后用总是好事情

	int i=blockIdx.x*(*threadNum)+threadIdx.x;
	double sum=0;
	int k=i*addNum;
	for(int j=0;j<addNum;j++)
	{
		sum+=a[k];
		k++;
	}
	res[i]=sum/addNum;
	__syncthreads();
	
}
__device__ double func0(double x)
{
	//被积分函数0,实际应用仅需修改此函数即可
	//同时考虑了代码复用性
	return sqrt(x+sqrt(x));
}
long*getCurrentTimeForDev()
{	
	long *time;
	hipMalloc(&time,sizeof(long));
	long *timenow=new long;
	*timenow=getCurrentTime();
	hipMemcpy(time,timenow,sizeof(long),hipMemcpyHostToDevice);
	return time;
}
double *DevValueD(double v,int len)//把host值转化为dev指针值
{
	double*res;
	hipMalloc(&res,sizeof(double)*len);
	double *val=new double[len];
	for(int i=0;i<len;i++)
		val[i]=v;
	hipMemcpy(res,val,sizeof(double)*len,hipMemcpyHostToDevice);
	return res;
}
int *DevValueI(int v,int len)
{
	int*res;
	hipMalloc(&res,sizeof(int)*len);
	int *val=new int[len];
	for(int i=0;i<len;i++)
		val[i]=v;
	hipMemcpy(res,val,sizeof(int)*len,hipMemcpyHostToDevice);
	return res;
}
#endif

double inte_cell_cpu(double l,double r)
{
	double x;
	double res=0;
	for(int i=0;i<Dev_Loop;i++)
	{
		x=(r-l)*randomf();
		res+=sqrt(x+sqrt(x*x));
	}
	res/=Dev_Loop;
	return res;
}


int work()
{
	int threadPerBlock=BlockN;
	int numBlocks= 256;
	size_t size = BlockN *numBlocks*sizeof(double);

	long st=getCurrentTime();

	hiprandState *state;
	hipMalloc(&state,sizeof(hiprandState)*1024*1024);//设立随机状态列

	double* d_A,*add_tem0,*add_tem1,*res;
	hipMalloc(&d_A, size);
	hipMalloc(&add_tem0, size/16);
	hipMalloc(&add_tem1, size/256);
	hipMalloc(&res,sizeof(double));

	inte_cell<<<numBlocks,threadPerBlock>>>(DevValueD(0.0,numBlocks*threadPerBlock),DevValueD(1.0,numBlocks*threadPerBlock),d_A,getCurrentTimeForDev(),state);
	/*
	big_plus<<<numBlocks/addNum,threadPerBlock>>>(d_A,add_tem0,DevValueI(1024,numBlocks*threadPerBlock));
	big_plus<<<numBlocks/addNum/addNum,threadPerBlock>>>(add_tem0,add_tem1,DevValueI(1024,numBlocks*threadPerBlock/addNum));
*/
	double *result=new double[numBlocks*threadPerBlock];
	/*
	FILE * out0,*out1,*out2;
	out0=fopen("data0.txt","w");
	out1=fopen("data1.txt","w");
	out2=fopen("data2.txt","w");


	cudaMemcpy(result, d_A, size, cudaMemcpyDeviceToHost);
	for(int i=0;i<1024;i++)
		fprintf(out0,"%f\n",result[i]);

	cudaMemcpy(result, add_tem0, size/addNum, cudaMemcpyDeviceToHost);
	for(int i=0;i<1024;i++)
		fprintf(out1,"%f\n",result[i]);
	*/
	/*
	for(int i=0;i<numBlocks*threadPerBlock;i++)
		fprintf(out0,"%f\n",result[i]);
		*/
	double fin_res=0;
	hipMemcpy(result,d_A, size, hipMemcpyDeviceToHost);
	for(int i=0;i<numBlocks*threadPerBlock;i++)
	{
		fin_res+=result[i];
	}
	fin_res/=(numBlocks*threadPerBlock);
	long ed=getCurrentTime();

	printf("GPU running Time:%ld\n",ed-st);
	printf("final:%16.14f\n",fin_res);

	/*
	st=getCurrentTime();
	double sum=0;
	for(int i=0;i<256;i++)
		for(int j=0;j<1024;j++)
		{
			sum+=inte_cell_cpu(0,1);
		}
	sum/=(1024*256);
	ed=getCurrentTime();
	printf("cpu:time:%ld,res:%15f\n",ed-st,sum);
*/
	/*
	fclose(out0);
	fclose(out1);
	fclose(out2);
	*/
	hipFree(d_A);
	hipFree(d_A);
	hipFree(add_tem0);
	hipFree(add_tem1);
	hipFree(state);
	hipFree(res);
}
int main()
{
	work();
}
