#include "hip/hip_runtime.h"
#include"stdio.h"
#include"stdlib.h"
#include<math.h>
#include <sys/time.h>    
#define USECUDA 1
#include<hip/hip_runtime.h>

#define Dev_Loop 1024
#define BlockN 1024
#define addNum 16
//这个程序由CUDA并行构架编写而成
//如果CUDA构架不够了解师兄也可以只看我的host代码部分,只需要把USECUDA 改为 0即可
long getCurrentTime()  
{  
	struct timeval tv;  
	gettimeofday(&tv,NULL);  
	return tv.tv_sec * 1000000 + tv.tv_usec;  
}
__device__ double randomf()//c 的随机数产生器改为产生小数
{
	return (double)(rand()/RAND_MAX);
}
#if USECUDA==1
//以下代码全部在CUDA构架下运行的
__global__ void inte_cell(double l,double r,double *res,double func(double))
{
	int i = blockIdx.x*BlockN+threadIdx.x;
	double x,sum=0;
	res[i]=0;
	for(int i=0;i<Dev_Loop;i++)
	{
		x=(r-l)*randomf();
		sum+=func(x);
	}
	res[i]=sum/Dev_Loop;
	__syncthreads();

}
__global__ void big_plus(double*a,double *res,int threadNum)
{
	//为了尽可能的利用并行效率，加法采用两次树形相加的形式，每次加addNum个
	//如此可以对付2^30次的快速相加
	//虽然嘛。。。。这是毫无意义的啦！因为本程序只有2^20次的相加
	//不过留个接口以后用总是好事情
	int i=blockIdx.x*threadNum+threadIdx.x;
	double sum=0;
	int k=i*addNum;
	for(int j=0;j<addNum;j++)
	{
		sum+=a[k];
		k++;
	}
	res[i]=sum/addNum;
	
}
__global__ void final_plus(double *a,double *res,int r)
{
	double s=0;
	for(int i=0;i<r;i++)
	{
		s+=a[i];
	}
	*res=s/r;
}
__device__ double func0(double x)
{
	//被积分函数0,实际应用仅需修改此函数即可
	//同时考虑了代码复用性
	return sqrt(x+sqrt(x));
}
#endif

double inte_cell_cpu(double l,double r,double func(double))
{
	double x;
	double res=0;
	for(int i=0;i<Dev_Loop;i++)
	{
		x=(r-l)*randomf();
		res+=func(x);
	}
	res/=Dev_Loop;
	return res;
}


int main()
{
	int threadPerBlock=1024;
	int numBlocks= BlockN;
	size_t size = 1024 *1024*1024* sizeof(double);
	long st=getCurrentTime();
	double* d_A,*add_tem0,*add_tem1,*res;
	hipMalloc(&d_A, size);
	hipMalloc(&add_tem0, size/16);
	hipMalloc(&add_tem1, size/256);
	hipMalloc(&res,sizeof(double));
	inte_cell<<<threadPerBlock,numBlocks>>>(0,1,d_A,func0);
	big_plus<<<1024/addNum,1024>>>(d_A,add_tem0,1024);
	big_plus<<<1024/addNum/addNum,1024>>>(add_tem0,add_tem1,1024);
	final_plus<<<1,1>>>(add_tem1, res ,1024*1024/addNum/addNum);
	double *result=new double;
	hipMemcpy(res, result, size, hipMemcpyDeviceToHost);
	long ed=getCurrentTime();
	
}
