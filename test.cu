#include"stdio.h"
#include<hip/hip_runtime.h>
#include <sys/time.h>    

#define N 1024
// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
	int i = threadIdx.x;
	for(int j=0;j<1000;j++)
		C[i] = (A[i] * B[i]);
}

long getCurrentTime()  
{  
	struct timeval tv;  
	gettimeofday(&tv,NULL);  
	return tv.tv_sec * 1000000 + tv.tv_usec;  
}
void cpu_VecAdd(int i,float* A, float* B, float* C)
{
	for(int j=0;j<1000;j++)
		C[i] = (A[i] * B[i]);
}
int main()
{
	// Kernel invocation with N threads
	printf("Hello,World\n");
	float *A=new float[N],*B=new float[N],*C=new float[N];
	for(int i=0;i<N;i++)
	{
		A[i]=i;
		B[i]=2*i;
	}

	size_t size = N * sizeof(float);

	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);
	float *e=new float[N];
	long st=getCurrentTime();
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	VecAdd<<<1, N>>>(d_A, d_B, d_C);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	long ed=getCurrentTime();
	printf("gpu running time:%ld\n",ed-st);
	st=getCurrentTime();
	for(int i=0;i<N;i++)
		cpu_VecAdd(i,A,B,e);
	ed=getCurrentTime();
	printf("cpu running time:%ld\n",ed-st);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	for(int i=0;i<N;i++)
	{
		//printf("%f ",C[i]);
	}
	printf("\n");
}
