#include"stdio.h"
#include<hip/hip_runtime.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include <sys/time.h>    

#define N 1024
// Kernel definition
__global__ void random_gpu(long* C,long* time,hiprandState*state)
{
	long i = threadIdx.x;
	long seed=(*time)*(i+1);//因为所有给定时间一定，所以我们只能通过对时间进行简单处理
	int offset=0;//完全独立的序列，所以offset全部为零来节约时间
	hiprand_init (seed,i,offset,&state[i]);//设置第i个随机序列
	C[i]=hiprand(&state[i]);//获得第i个随机序列的随机值
}

long getCurrentTime()  
{  
	struct timeval tv;  
	gettimeofday(&tv,NULL);  
	return tv.tv_sec * 1000000 + tv.tv_usec;  
}
long*getCurrentTimeForDev()
{	long *time;
	hipMalloc(&time,sizeof(long));
	long *timenow=new long;
	*timenow=getCurrentTime();
	hipMemcpy(time,timenow,sizeof(long),hipMemcpyHostToDevice);
	return time;
}
int main()
{
	size_t size = N * sizeof(float);

	long* C=new long[N];
	long st=getCurrentTime();
	hiprandState *state;
	long *d_C;
	hipMalloc(&state,sizeof(hiprandState)*N);//设立随机状态列
	hipMalloc(&d_C, size);
	random_gpu<<<1,N>>>(d_C,getCurrentTimeForDev(),state);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	long ed=getCurrentTime();
	printf("gpu running time:%ld\n",ed-st);
	hipFree(d_C);
	for(int i=0;i<10;i++)
	{
		printf("%ld ",C[i]);
	}
	delete[] C;
	printf("\n");
}
