#include"stdio.h"
#include<hip/hip_runtime.h>
#include <sys/time.h>    

#define len 1
#define WIDTH 128
// Kernel definition
__device__ float& getPos(float *T,int x,int y,int w)
{
	return *(T+y*w+x);
}
// 处理：正方形，二维热流场
//          dN
//      dW  dT  dE
//		    dS
__global__ void Calc_Cell(float* T0)
{
	float dW,dE,dN,dS,dT;
	int x=blockIdx.x+1;
	int y=threadIdx.x+1;
	int w=WIDTH;
	for(int i=0;i<100000;i++)
	{
		dN=(getPos(T0,x,y,w)-getPos(T0,x,y-1,w))/len;
		dS=(getPos(T0,x,y+1,w)-getPos(T0,x,y,w))/len;
		dW=(getPos(T0,x,y,w)-getPos(T0,x-1,y,w))/len;
		dE=(getPos(T0,x+1,y,w)-getPos(T0,x,y,w))/len;
		dT=((dS-dN)/len+(dE-dW)/len)*0.1;
		__syncthreads();
		getPos(T0,x,y,w)=getPos(T0,x,y,w)+dT;
	}
}

//储存是行主序，然而坐标是列主序
long getCurrentTime()  
{  
	struct timeval tv;  
	gettimeofday(&tv,NULL);  
	return tv.tv_sec * 1000000 + tv.tv_usec;  
}

int main()
{
	FILE *fp=fopen("a.txt","w");
	size_t size=128*WIDTH*sizeof(float);
	float*d_A;
	hipMalloc(&d_A, size);
	float A[128*WIDTH]={0};
	for(int i=0;i<128;i++)
		A[i]=100;
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	//在这个计算中我们给定边界条件，仅仅考虑内部的运行状态
	Calc_Cell<<<126,126>>>(d_A);
	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
	for(int i=0;i<128;i++)
	{
		for(int j=0;j<WIDTH;j++)
		{
			fprintf(fp,"%d %d %f \n",i,j,A[i*WIDTH+j]);
		}
	}
	hipFree(d_A);
}
